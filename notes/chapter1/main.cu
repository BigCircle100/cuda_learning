
#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel func
__global__
void square(float *d_out, float *d_in){
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f*f;
}

int main(int argc, char **argv){
  const int ARRAY_SIZE = 64;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  float h_in[ARRAY_SIZE];
  for(int i = 0; i < ARRAY_SIZE; i++){
    h_in[i] = float(i);
  }
  float h_out[ARRAY_SIZE];

  float * d_in;
  float * d_out;

  // alloc device mem, use bytes
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // s2d
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // <<<block, thread>>>
  square<<<1, ARRAY_SIZE>>>(d_out, d_in);

  // d2s
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  for (int i = 0 ; i < ARRAY_SIZE; i++){
    printf("%f", h_out[i]);
    printf(((i%4)!=3)?"\t":"\n");
  }

  // delete
  hipFree(d_in);
  hipFree(d_out);

  return 0;

  
}