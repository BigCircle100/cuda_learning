
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <algorithm>
#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


// 有个地方需要注意，&的优先级低于==

__global__
void histogram_kernel(const int pass, 
                      int * d_bins,
                      const int * const d_in,
                      const int size){
  int mid = threadIdx.x + blockDim.x * blockIdx.x;
  if (mid >= size)
    return;
  int one = 1;
  int bin = (d_in[mid] & (one << pass)) == (one << pass);
  if (bin)
    atomicAdd(&d_bins[1], 1);
  else
    atomicAdd(&d_bins[0], 1);
}

__global__
void exclusive_scan_kernel(const int pass,
                            const int *const d_in,
                            int *d_scan,
                            const int size){
  int mid = threadIdx.x + blockDim.x * blockIdx.x;
  if (mid >= size)
    return;
  int val = 0;
  int one = 1;
  // 这里记录的值是mid-1的值，而不是mid，相当于把inclusive scan改成了exclusive scan
  // 也就是此时得到的d_out数组（pred）在前面多了一位0
  // 最后一位不重要，因此不需要改size，最后一位直接不要了
  // 举例：
  // 0 1 2 3 4    -》原数组
  // 0 1 0 1 0    -》正常情况下，二进制数值最后一位是1时对应的pred
  // 0 0 1 0 1    -》上述pred前面添加一位0的新pred
  // 0 0 1 1 2    -》对上面pred进行inclusive scan，和对原pred进行exclusive scan效果相同
  if (mid > 0)
    val = ((d_in[mid-1] & (one << pass)) == (one << pass)) ? 1 : 0;
  
  d_scan[mid] = val;

  __syncthreads();


  // 注意：__syncthreads()同步操作不要放到核函数的if条件语句当中，因为不是所有线程都能到达这个位置。否则可能导致死锁或同步错位的未定义行为。
  for (int s = 1; s <= size; s *=2){
    if (mid - s >= 0){
      val = d_scan[mid-s];
    }
    __syncthreads();
    if (mid - s >= 0){
      d_scan[mid] += val;
    }
    __syncthreads();
  }
}

__global__
void move_kernel( const int pass,
                  const int *const d_in,
                  int *d_out,
                  int *d_scan,
                  int one_pos,
                  const int size){
  int mid = threadIdx.x + blockDim.x * blockIdx.x;
  if (mid >= size)
    return;
  int scan = 0;
  int base = 0;
  int one = 1;
  if ((d_in[mid] & (one << pass)) == (one << pass)){
    base = one_pos;
    scan = d_scan[mid];
  }else{
    base = 0;
    scan = mid - d_scan[mid];
  }
  d_out[base+scan] = d_in[mid];

}

int main(){
  int length = 31;
  int h_in[length];
  int h_out[length];
  int h_bins[2];
  for (int i = 0; i < length; i++){
    h_in[i] = rand();
    // h_in[i] = i;
  }

  printf("h_in: \n");
  for (int i = 0; i < length; i++){
    printf("%d ", h_in[i]);
  }
  printf("\n");

  int *d_in, *d_out, *d_scan, *d_bins;
  hipMalloc((void**)&d_in, sizeof(int)*length);
  hipMalloc((void**)&d_out, sizeof(int)*length);
  hipMalloc((void**)&d_scan, sizeof(int)*length);
  hipMalloc((void**)&d_bins, sizeof(int)*2);

  hipMemcpy(d_in, h_in, sizeof(int)*length, hipMemcpyHostToDevice);


  int block_width = 32;
  dim3 thread_dim(block_width);
  dim3 block_dim(1);

  for (int pass = 0; pass < 32; ++pass){
    hipMemset(d_bins, 0, 2*sizeof(int));
    hipMemset(d_scan, 0, length*sizeof(int));
    histogram_kernel<<<block_dim, thread_dim>>>(pass, d_bins, d_in, length);
    hipDeviceSynchronize();
    hipMemcpy(h_bins, d_bins, 2*sizeof(int), hipMemcpyDeviceToHost);
    exclusive_scan_kernel<<<block_dim, thread_dim>>>(pass, d_in, d_scan, length);
    hipDeviceSynchronize();
    move_kernel<<<block_dim, thread_dim>>>(pass, d_in, d_out, d_scan, h_bins[0], length);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, sizeof(int)*length, hipMemcpyDeviceToHost);
    hipMemcpy(d_in, d_out, sizeof(int)*length, hipMemcpyDeviceToDevice);
  }


  hipMemcpy(h_out, d_out, sizeof(int)*length, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_bins);
  hipFree(d_scan);

  printf("h_out: \n");
  for (int i = 0; i < length; i++){
    printf("%d ", h_out[i]);
  }
  printf("\n");

  std::sort(h_in, h_in+length);
  bool is_equal = true;
  for (int i = 0; i < length; i++){
    if (h_in[i] != h_out[i]){
      is_equal = false;
      break;
    }
  }
  if (is_equal){
    printf("correct anwser\n");
  }else{
    printf("wrong anwser\n");
  }

}